#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int recursiveReduce(int *data,int const size)
{
	if (size==1) return data[0];
	int const stride = size/2;
	for(int i =0;i<stride;i++)
	{
		data[i] += data[i+stride];
	}
	return recursiveReduce(data,stride);
}

//Neighbored pair implementation with divergence
__global__ void reduceNeighbored(int *g_idata,int *g_odata,unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

	int *idata = g_idata + blockIdx.x * blockDim.x;

	if(idx >= n) return;

	for(int stride = 1;stride < blockDim.x; stride *= 2)
	{
		if(tid % (2 * stride) == 0)
		{
			idata[tid] += idata[tid + stride];
		}

		__syncthreads();
	}
	if(tid ==0)g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	 int *idata = g_idata + blockIdx.x * blockDim.x;

	 if(idx >= n)return;

	 for(int stride = 1;stride < blockDim.x;stride*=2)
	 {
		 int index = 2 * stride * tid;//rearrange consecutive threads
		 if(index < blockDim.x)
		 {
			 idata[index] += idata[index + stride];
		 }

		 __syncthreads();
	 }
	 if(tid==0)g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

	int *idata = g_idata + blockIdx.x * blockDim.x;

	if(idx >= n)return;
	for(int stride = blockDim.x / 2;stride > 0;stride/=2)
	{
		if(tid < stride)
		{
			idata[tid] += idata[tid + stride];
		}
		__syncthreads();
	}

	if(tid==0)g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x * 2;

	int *idata = g_idata + blockIdx.x * blockDim.x * 2;

	if(idx + blockDim.x < n) g_idata[idx] += g_idata[idx + blockDim.x];
	__syncthreads();

	for(int stride = blockDim.x / 2;stride > 0;stride >>=1)
	{
		if(tid < stride)
		{
			idata[tid] += idata[tid + stride];
		}
		__syncthreads();
	}

	if(tid==0)g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrolling4(int *g_idata, int *g_odata, unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x * 4;

	int *idata = g_idata + blockIdx.x * blockDim.x * 4;

	if(idx + 3 * blockDim.x < n)
	{
		g_idata[idx] += g_idata[idx + blockDim.x];
		g_idata[idx] += g_idata[idx + 2*blockDim.x];
		g_idata[idx] += g_idata[idx + 3*blockDim.x];
	}

	__syncthreads();

	for(int stride = blockDim.x / 2; stride > 0;stride /= 2)
	{
		if(tid < stride)
		{
			idata[tid] += idata[tid + stride];
		}

		__syncthreads();
	}
	if(tid == 0)g_odata[blockIdx.x] = idata[0];
}

int main(int argc,char **argv)
{
	bool result = false;
	double iStart,iElaps;
	int gpu_sum = 0;
	int size = 1<<14;
	printf("array size %d \n",size);
	int blocksize = 512;
	if(argc > 1) blocksize = atoi(argv[1]);
	dim3 block(blocksize,1);
	dim3 grid((size + block.x - 1) / block.x,1);

	//allocate host memory
	size_t nBytes = size * sizeof(int);
	int *h_idata = (int *)malloc(nBytes);
	int *h_odata = (int *)malloc(grid.x * sizeof(int));
	int *tmp = (int *)malloc(nBytes);

	//initialize the array
	for(int i=0;i<size;i++)
	{
		h_idata[i] = (int)( rand() & 0xFF );
	}
	memcpy(tmp,h_idata,nBytes);
	
	//allocate device memory

	int *d_idata = NULL;
	int *d_odata = NULL;
	hipMalloc((void **)&d_idata,nBytes);
	hipMalloc((void **)&d_odata,grid.x * sizeof(int));

	// cpu reduction
    	iStart = seconds();
    	int cpu_sum = recursiveReduce (tmp, size);
    	iElaps = seconds() - iStart;
    	printf("cpu reduce      elapsed %f sec cpu_sum: %d\n", iElaps, cpu_sum);

    	// kernel 1: reduceNeighbored
    	hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
    	hipDeviceSynchronize();
    	iStart = seconds();
    	reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    	hipDeviceSynchronize();
    	iElaps = seconds() - iStart;
    	hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),hipMemcpyDeviceToHost);
    	gpu_sum = 0;
    	for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];
    	printf("gpu Neighbored elapsed %f sec gpu_sum: %d <<<grid %d block ""%d>>>\n", iElaps, gpu_sum, grid.x, block.x);

	// kernel 2: reduceNeighboredLess
        hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        iStart = seconds();
        reduceNeighboredLess<<<grid, block>>>(d_idata, d_odata, size);
        hipDeviceSynchronize();
        iElaps = seconds() - iStart;
        hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),hipMemcpyDeviceToHost);
        gpu_sum = 0;
        for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];
        printf("gpu NeighboredLess elapsed %f sec gpu_sum: %d <<<grid %d block ""%d>>>\n", iElaps, gpu_sum, grid.x, block.x);

	// kernel 3: reduceInterleaved
        hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        iStart = seconds();
        reduceInterleaved<<<grid, block>>>(d_idata, d_odata, size);
        hipDeviceSynchronize();
        iElaps = seconds() - iStart;
        hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),hipMemcpyDeviceToHost);
        gpu_sum = 0;
        for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];
        printf("gpu reduceInterleaved elapsed %f sec gpu_sum: %d <<<grid %d block ""%d>>>\n", iElaps, gpu_sum, grid.x, block.x);
	
	// kernel 4:reduceUnrolling2
	hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	iStart = seconds();
	reduceUnrolling2<<< grid.x/2, block >>>(d_idata, d_odata, size);
	hipDeviceSynchronize();
	iElaps = seconds() - iStart;
	hipMemcpy(h_odata, d_odata, grid.x/2*sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x / 2; i++) gpu_sum += h_odata[i];
	printf("gpu Unrolling2 elapsed %f sec gpu_sum: %d <<<grid %d block %d>>>\n",iElaps,gpu_sum,grid.x/2,block.x);
	
	// kernel 5:reduceUnrolling4
        hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        iStart = seconds();
        reduceUnrolling4<<< grid.x/4, block >>>(d_idata, d_odata, size);
        hipDeviceSynchronize();
        iElaps = seconds() - iStart;
        hipMemcpy(h_odata, d_odata, grid.x/4*sizeof(int), hipMemcpyDeviceToHost);
        gpu_sum = 0;
        for (int i = 0; i < grid.x / 4; i++) gpu_sum += h_odata[i];
        printf("gpu Unrolling4 elapsed %f sec gpu_sum: %d <<<grid %d block %d>>>\n",iElaps,gpu_sum,grid.x/4,block.x);
	// free host memory
    	free(h_idata);
    	free(h_odata);
    	free(tmp);

    	// free device memory
    	hipFree(d_idata);
    	hipFree(d_odata);

    	// reset device
    	hipDeviceReset();

    	// check the results
    	result = (gpu_sum == cpu_sum); // this will only check the last kernel result

    	if(!result) printf("Test failed!\n");

    	return EXIT_SUCCESS;
}



